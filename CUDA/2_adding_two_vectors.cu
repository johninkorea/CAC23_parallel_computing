
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// CUDA kernel to add two vectors
__global__ void addVectorsCUDA(const int* vec1, const int* vec2, int* result, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        result[index] = vec1[index] + vec2[index];
    }
}

int main() {
    std::vector<int> vector1 = {1, 2, 3, 4, 5};
    std::vector<int> vector2 = {6, 7, 8, 9, 10};

    int size = vector1.size();
    std::vector<int> sum(size);

    // Allocate memory on the GPU
    int* deviceVec1;
    int* deviceVec2;
    int* deviceResult;
    hipMalloc(&deviceVec1, size * sizeof(int));
    hipMalloc(&deviceVec2, size * sizeof(int));
    hipMalloc(&deviceResult, size * sizeof(int));

    // Copy input vectors from host to device
    hipMemcpy(deviceVec1, vector1.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceVec2, vector2.data(), size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel on the GPU
    int threadsPerBlock = 256;
    int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    addVectorsCUDA<<<numBlocks, threadsPerBlock>>>(deviceVec1, deviceVec2, deviceResult, size);

    // Copy result from device to host
    hipMemcpy(sum.data(), deviceResult, size * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(deviceVec1);
    hipFree(deviceVec2);
    hipFree(deviceResult);

    std::cout << "Result: ";
    for (const auto& value : sum) {
        std::cout << value << " ";
    }
    std::cout << std::endl;

    return 0;
}

