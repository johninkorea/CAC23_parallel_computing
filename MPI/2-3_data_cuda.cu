
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <chrono>

using namespace std;
using namespace chrono;
using std::cout; using std::endl;

__device__ double f(double x) {
    return pow(x, 2) - 3 * x + 2;
}

__global__ void calculateFunction(double start, double step, double *results, int numSteps) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numSteps) {
        double x = start + tid * step;
        results[tid] = f(x);
    }
}

int main() {
    system_clock::time_point start_time = system_clock::now();
    
    clock_t start1, end1;
    start1 = clock();

    double start = 0.0;
    double end = 10000.0;
    double step = 0.001;
    int numSteps = static_cast<int>((end - start) / step);

    // Allocate memory on the host
    double* hostResults = new double[numSteps];

    // Allocate memory on the device
    double* deviceResults;
    hipMalloc((void**)&deviceResults, numSteps * sizeof(double));

    // Copy data from host to device
    hipMemcpyToSymbol(HIP_SYMBOL("start"), &start, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL("step"), &step, sizeof(double));

    // Launch kernel
    int blockSize = 10000;
    // int gridSize = (numSteps + blockSize - 1) / blockSize;
    int gridSize = 100;
    calculateFunction<<<gridSize, blockSize>>>(start, step, deviceResults, numSteps);

    // Copy results from device to host
    hipMemcpy(hostResults, deviceResults, numSteps * sizeof(double), hipMemcpyDeviceToHost);

    // Print results
    // for (int i = 0; i < numSteps; ++i) {
    //     double x = start + i * step;
    //     cout << "f(" << x << ") = " << hostResults[i] << endl;
    // }

    // Clean up
    delete[] hostResults;
    hipFree(deviceResults);
    
    system_clock::time_point end_time = system_clock::now();
    end1 = clock();
    // nanoseconds nano = end1 - start1;
    printf("%f\n",((float)end1) / CLOCKS_PER_SEC * 1000000);
    
    return 0;
}
